#include <iostream>
#include <cassert>
#include <string>
#include <vector>
// #include "utils.h"
#include "../include/utils.h"
#include "../include/lin_alg.h"


float* transferMatrixToDevice(float **matrix, int height, int width) {
    float* deviceMatrix;
    hipMalloc(&deviceMatrix, height*width*sizeof(float));
    for(int i = 0; i < height; i++) {
        hipMemcpy(deviceMatrix+(i*width), matrix[i], sizeof(float)*width, hipMemcpyHostToDevice);
    }
    return deviceMatrix;
}

int main(int argc, char** argv) {
    int numClasses = 10;
    std::cout << "Hello World!" << std::endl;
    std::cout << "Train data path: " << argv[1] << std::endl;
    if(argc != 3) {
        std::cout << "Need to specify paths for loading in the training data and the training labels" << std::endl;
        return 0;
    }
    std::string train_data_path = argv[1];
    std::string train_label_path = argv[2];
    std::cout << "args" << std::endl;
    std::vector<std::vector<int>> inputs = readDataFromUByteFile(train_data_path);
    int size = inputs.size();
    int nFeatures = inputs[0].size();
    // std::cout << height << " " << width << std::endl;
    std::vector<float> input(size*nFeatures, 0.0f);
    for(int i = 0; i < size; i++) {
        for(int j = 0; j < nFeatures; j++) {
            input[i*nFeatures + j] = (float) inputs[i][j] / 255.0;
        }
    }
    std::vector<std::vector<int>> outputs = readDataFromUByteFile(train_label_path);
    std::vector<float> weights = initializeRandomArray(nFeatures, numClasses);
    std::vector<float> product(size*nFeatures*10, 0.0);
    float learning_rate = 0.005;
    int nWorkers = 4;
    int nThreadsPerWorker = 2;
    int BATCH_SIZE = size / (nWorkers * nThreadsPerWorker);

    float ** output;
    output = (float**)malloc(sizeof(float*) * size);
    for (int i = 0; i < size; i++) {
        output[i] = (float *)malloc(10*sizeof(float));
        for(int j = 0; j < 10; j++) {
            output[i][j] = 0;
        }
        output[i][outputs[i][0]] = 1.0;
    }
    float *d_outputs = transferMatrixToDevice(output, size, 10);


    //declare device variables
    float *d_inputs;
    float *d_weights;
    float *d_product;
    hipMalloc(&d_inputs, size*nFeatures*sizeof(float));
    hipMemcpy(d_inputs, input.data(), size*nFeatures*sizeof(float), hipMemcpyHostToDevice);
    hipMalloc(&d_weights, nFeatures*numClasses*sizeof(float));
    hipMemcpy(d_weights, weights.data(), nFeatures*numClasses*sizeof(float), hipMemcpyHostToDevice);
    hipMalloc(&d_product, size*numClasses*sizeof(float));
    hipMemcpy(d_product, product.data(), size*numClasses*sizeof(float), hipMemcpyHostToDevice);
    forward_pass<<<nWorkers, nThreadsPerWorker>>>(d_inputs, d_weights, d_outputs, d_product, BATCH_SIZE, nFeatures, numClasses);
    hipDeviceSynchronize();

    float *produce = (float*)malloc(size*numClasses*sizeof(float));
    hipMemcpy(produce, d_product, size*numClasses*sizeof(float), hipMemcpyDeviceToHost);
    printMatrix(produce, size, numClasses);
    hipFree(d_inputs);
    hipFree(d_weights);
    hipFree(d_product);

    return 0;
}