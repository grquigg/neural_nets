#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include "../include/lin_alg.h"

//////////DEVICES////////

__device__ void softmax(float* product, int product_height, int product_width) {
    float total = 0.0;
    float logSumTotal = 0.0;
    for (int i = 0; i < product_height; i++) {
        total = 0.0;
        for (int j = 0; j < product_width; j++) {
            total += exp(product[i*product_width+j]);
        }
        logSumTotal = log(total);
        float prob_sums = 0.0;
        for (int j = 0; j < product_width; j++) {
            product[i*product_width+j] = exp(product[i*product_width+j] - logSumTotal);
            prob_sums += product[i*product_width+j];
        }

    }
}

__device__ void sigmoid(float* inputs, int size) {
    for(int i = 0; i < size; i++) {
        inputs[i] = (1/ (1+expf(-inputs[i])));
    }
}
__global__ void sigmoidD(float* activations, int height, int width, float * delta) {
    if((height * width) % (gridDim.x * blockDim.x) != 0) {
        printf("Bad outcome\n");
    }
    printf("Sigmoid\n");
    int batch = (height * width) / (gridDim.x * blockDim.x);
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    for(int i = 0; i < batch; i++) {
        delta[index*batch+i] *= activations[index*batch+i] * (1-activations[index*batch+i]);
    }
}
__device__ float* transposeMatrix(float * matrix, int matrix_height, int matrix_width) {
    float * transpose = new float[matrix_width*matrix_height];
    for(int i = 0; i < matrix_height; i++) {
        for(int j = 0; j < matrix_width; j++) {
            transpose[i*matrix_width+j] = matrix[j*matrix_height+i];
            // printf("Valid %d %d %f %f\n", i, j, transpose[i*matrix_width+j], matrix[j*matrix_height+i]);
        }
    }
    return transpose;
}

__device__ void dotProduct(float* inputs, float* weights, float * product, int vector_h, int vector_w, int weight_h, int weight_w) {
    //initialize the matrix
    //dot product is ALWAYS computed as the rows of the first matrix by the columns of the second matrix
    if (vector_w != weight_h) {
        printf("invalid values\n");
        return;
    }
    for(int i = 0; i < vector_h; i++) { //for every row in the first matrix
        for(int j = 0; j < weight_w; j++) { //for every column in the second matrix
            product[i*weight_w+j] = 0.0;
            for(int k = 0; k < vector_w; k++) { //we compute the kth entry in row i of the INPUTS times the kth entry in column j of the WEIGHTS
                product[i*weight_w+j] += inputs[i*vector_w+k] * weights[k*weight_w+j];
                // printf("This %d %d %f %f\n", i, j, inputs[i*vector_w+k], weights[k*weight_w+j]);
            }
            printf("%f\n", product[i*weight_w+j]);
        }
    }
}

__device__ void dotProduct(float* inputs, float* weights, float * product, int vector_h, int vector_w, int weight_h, int weight_w, float* bias) {
    if (vector_w != weight_h) {
        printf("invalid values\n");
        return;
    }
    for(int i = 0; i < vector_h; i++) { //for every row in the first matrix
        for(int j = 0; j < weight_w; j++) { //for every column in the second matrix
            product[i*weight_w+j] = 0.0;
            for(int k = 0; k < vector_w; k++) { //we compute the kth entry in row i of the INPUTS times the kth entry in column j of the WEIGHTS
                product[i*weight_w+j] += inputs[i*vector_w+k] * weights[k*weight_w+j];

            }
            product[i*weight_w+j] += bias[j];
        }
    }
}
__global__ void dotProductTransposeSegmented(float* inputs, float* weights, float * product, int vector_h, int vector_w, int weight_h, int weight_w, bool useHeight) {
    if(useHeight && vector_h == weight_h) {
        if(vector_w % (gridDim.x * gridDim.y) != 0 || weight_w % (blockDim.x * blockDim.y) != 0) {
            printf("BAD RESULT\n");
            return;
        }
        printf("Vector w %d weight w %d\n", vector_w, weight_w);
        int batch_size_x = vector_w / (gridDim.x);
        int batch_size_y = weight_w / (blockDim.x);
        int index_x = blockIdx.x;
        int index_y = threadIdx.x;
        printf("Batch size x %d Batch size y %d\n", batch_size_x, batch_size_y);
        printf("Index x %d, Index y %d\n", index_x, index_y);
        //index_x*batch_size_x indicates the starting row of the input matrix
        //index_x*batch_size_x*weight_x indicates the starting row of the product matrix
        //index_y*batch_size_y indicates the starting column of the weight matrix
        for(int i = 0; i < batch_size_x; i++) { //
            for(int j = 0; j < batch_size_y; j++) {
                printf("starting column in inputs: %d\nstarting column in weights: %d\n\n", index_x*batch_size_x+i, index_y*batch_size_y+j);
                product[(index_x*batch_size_x+i)*weight_w+(index_y*batch_size_y+j)] = 0.0f;
                for(int k = 0; k < vector_h; k++) {
                    product[(index_x*batch_size_x+i)*weight_w+(index_y*batch_size_y+j)] += inputs[index_x*batch_size_x+i+(vector_w*k)] * weights[index_y*batch_size_y+j+(weight_w*k)];
                }
                printf("%f\n", product[(index_x*batch_size_x+i)*weight_w+(index_y*batch_size_y+j)]);
            }
        }
    } else if(vector_w == weight_w) {
        if(vector_h % (gridDim.x * gridDim.y) != 0 || weight_h % (blockDim.x * blockDim.y) != 0) {
            printf("BAD RESULT\n");
            return;
        }
        int batch_size_x = vector_h / (gridDim.x*gridDim.y);
        int batch_size_y = weight_h / (blockDim.x*blockDim.y);
        // printf("Batch size x %d batch size y %d\n", batch_size_x, batch_size_y);
        int index_x = blockIdx.x*gridDim.y + blockIdx.y;
        int index_y = threadIdx.x*blockDim.y + threadIdx.y;
        // printf("Index x %d, Index y %d\n", index_x, index_y);
        for(int i = 0; i < batch_size_x; i++) {
            for(int j = 0; j < batch_size_y; j++) {
                printf("Index at x %d index at y %d, Start position in product array %d\n", index_x*batch_size_x+i, index_y*batch_size_y+j, (index_x*batch_size_x+i)*weight_h+index_y*batch_size_y+j);
                product[(index_x*batch_size_x+i)*weight_h+index_y*batch_size_y+j] = 0.0f;
                for(int k = 0; k < vector_w; k++) {
                    product[(index_x*batch_size_x+i)*weight_h+index_y*batch_size_y+j] += inputs[(index_x*batch_size_x+i)*vector_w+k] * weights[(index_y*batch_size_y+j)*weight_w+k];
                }
                printf("%f\n", product[(index_x*batch_size_x+i)*weight_h+index_y*batch_size_y+j]);
            }
        }
    }
}
__device__ void dotProductTranspose(float* inputs, float* weights, float * product, int vector_h, int vector_w, int weight_h, int weight_w) {
    //remember that we want the resulting matrix to be of shape [vector_h, weight_w]
    if(vector_h == weight_h) {
    for(int i = 0; i < vector_w; i++) {
        for(int j = 0; j < weight_w; j++) {
            product[i*weight_w+j] = 0.0;
            for(int k = 0; k < vector_h; k++) {
                product[i*weight_w+j] += inputs[k*vector_w+i] * weights[k*weight_w+j];
            }
        }
    }
    } else if(vector_w == weight_w) {
        for(int i = 0; i < vector_h; i++) {
            for(int j = 0; j < weight_h; j++) {
                product[i*weight_h+j] = 0.0;
                for(int k = 0; k < vector_w; k++) {
                    product[i*weight_h+j] += inputs[i*vector_w+k] * weights[j*weight_w+k];
                }
            }
        }
    } else {
        printf("INVALID DIMS FOR DOT PRODUCT\n");
    }
}

__global__ void matrixSubtract(float * matrix1, float *matrix2, int m1_h, int m1_w, int m2_h, int m2_w, float* outVec) {
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    if((m1_h*m1_w) % (gridDim.x *blockDim.x) != 0) {
        printf("BAD INPUT\n");
    }
    //this can be literally be flattened out to a linear operation
    int batch_size = (m1_h*m1_w)/(gridDim.x *blockDim.x);
    for(int i = 0; i < batch_size; i++) {
        outVec[index*batch_size+i] = matrix1[index*batch_size+i] - matrix2[index*batch_size+i];
    }
}

__device__ void matrixAdd(float * matrix1, float * matrix2, int m1_h, int m1_w) {
    for(int i = 0; i < m1_h; i++) {
        for(int j = 0; j < m1_w; j++) {
            matrix1[i*m1_w+j] += matrix2[i*m1_w+j];
        }
    }
}

__device__ void matrixMultiplyByScalar(float* mat, int m1_h, int m1_w, float scalar) {
    for(int i = 0; i < m1_h; i++) {
        for(int j = 0; j < m1_w; j++) {
            mat[(i*m1_w)+j]*= scalar;
        }
    }
}
//////////GLOBALS////////
/*
*gradients: the gradient vector
begin_part: the index of where this threads cumulative gradients begin in the subsection
end_part: the index of where this threads cumulative gradients end in the subsection
total_steps: the total number of steps that each thread needs to perform in order to acheive the full cumulative gradient
step_size: the total distance in grads that we need to jump every time
*/
__global__ void ringReduce(float* gradients, const int total_steps, const int step_size, const int chunk_size) {
    //we achieve our reduction in two loops: update and set
    //in the update loop, we're simply calculating the cumulative sum of each part of the respective gradients
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    int begin_part = index*chunk_size;
    int end_part = (index+1)*chunk_size;
    for(int i = 1; i < total_steps; i++) {
        for(int j = begin_part; j < end_part; j++) {
            gradients[j] += gradients[(i*step_size)+j];
        }
    }
}

__global__ void ringReduce(LogisticRegression * model, const int total_steps, const int step_size, const int chunk_size) {
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    int begin_part = index*chunk_size;
    int end_part = (index+1)*chunk_size;
    for(int i = 1; i < total_steps; i++) {
        for(int j = begin_part; j < end_part; j++) {
            model->gradients[j] += model->gradients[(i*step_size)+j];
        }
    }
    // printf("Ring reduce\n");
}
__global__ void predict(float * inputs, float* weights, float * product, int size, int n_features, int n_classes) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int batch = size / (blockDim.x * gridDim.x);
    dotProduct(inputs+(i*n_features*batch), weights, product+(i*n_classes*batch), batch, n_features, n_features, n_classes);
    softmax(product+(i*n_classes*batch), batch, n_classes);
}

__global__ void predict(LogisticRegression* model, float* inputs, float* product, int size) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int batch = size / (blockDim.x * gridDim.x);
    dotProduct(inputs+(i*(model->nFeatures)*batch), model->weights, product+(i*(model->nClasses)*batch), batch, model->nFeatures, model->nFeatures, model->nClasses);
    softmax(product+(i*(model->nClasses)*batch), batch, (model->nClasses));
}
// __global__ void forward_pass(float* inputs, float* weights, float* outputs, float* product, float* gradients, int size, int n_features, int n_classes) {
//     int i = blockIdx.x*blockDim.x + threadIdx.x;
//     int batch = size / (blockDim.x * gridDim.x);
//     matrixSubtract(product+(i*n_classes*batch), outputs+(i*n_classes*batch), batch, n_classes, batch, n_classes, product+(i*n_classes*batch));
//     dotProductTranspose(inputs+(i*batch*n_features), product+(i*batch*n_classes), gradients+(i*n_features*n_classes), batch, n_features, batch, n_classes);
// }

// __global__ void forward_pass(LogisticRegression* model, float* inputs, float* outputs, float* product, int size, int nClasses) {
//     int i = blockIdx.x*blockDim.x + threadIdx.x;
//     int batch = size / (blockDim.x * gridDim.x);
//     matrixSubtract(product+(i*nClasses*batch), outputs+(i*nClasses*batch), batch, nClasses, batch, nClasses, product+(i*nClasses*batch));
//     dotProductTranspose(inputs+(i*batch*(model->nFeatures)), product+(i*batch*(model->nClasses)), ((*model).gradients)+(i*(model->nClasses)*(model->nFeatures)), batch, (model->nFeatures), batch, (model->nClasses));
// }

// __global__ void backward_pass(float* weights, float * gradients, int batch_size, float learning_rate, int n_features, int n_classes) {
//     int i = blockIdx.x*blockDim.x + threadIdx.x;
//     //BATCH_SIZE*n_classes length vector
//     int batch = n_features / (blockDim.x * gridDim.x);
//     matrixMultiplyByScalar(gradients+(i*batch*n_classes), batch, n_classes, learning_rate/(float) batch_size);
//     matrixSubtract(weights+(i*n_classes*batch), gradients+(i*n_classes*batch), batch, n_classes, batch, n_classes, weights+(i*n_classes*batch));
// }

__global__ void backward_pass(LogisticRegression* model, int batch_size, float learning_rate) {
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    //BATCH_SIZE*n_classes length vector
    int batch = model->nFeatures / (blockDim.x * gridDim.x);
    int start = index*batch*(model->nClasses);
    for(int i = 0; i < batch; i++) {
        for(int j = 0; j < model->nClasses; j++) {
            (*model).gradients[start+i*(model->nClasses)+j] *= (learning_rate / batch_size);
            (*model).weights[start+i*(model->nClasses)+j] -=  (*model).gradients[start+i*(model->nClasses)+j];
        }
    }
    // printf("Finish backward\n");
}

///NEURAL NETWORK CODE

__global__ void predict(NeuralNetwork* model, float* inputs, float* product, int* offsets, int size) {
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    float items = size / (float) (blockDim.x * gridDim.x);
    int batch = ceil(items);
    if (index * batch >= size) {
        return;
    }
    /*
    Each thread takes a chunk of the input data and feeds it all the way through the neural network, storing the intermediary results in product along the way.
    Just going to use softmax as the default activation as I'm more familiar with how that works anyways
    */
    float* input = inputs+(index*(model->layer_size[0])*batch);
    int batch_size = min(size-(index*batch), batch);
    // printf("batch size: %d\n", batch_size);
    float* out = (product+(index*(model->layer_size[1])*batch));
    dotProduct(input, model->weights[0], out, batch_size, model->layer_size[0], model->layer_size[0], model->layer_size[1], model->biases[0]);
    sigmoid(out, batch_size*model->layer_size[1]);
    for(int i = 1; i < model->nLayers; i++) {
        input = out;
        out = product+offsets[i]+(index*(model->layer_size[i+1])*batch);
        dotProduct(input, model->weights[i], out, batch_size, model->layer_size[i], model->layer_size[i], model->layer_size[i+1], model->biases[i]);
        softmax(out, batch_size, (model->layer_size[i+1]));
    }
}

// __global__ void backprop(NeuralNetwork* model, float* inputs, float* outputs, float* activations, float* deltas, int * offsets, int size, int nClasses) {
//     int index = blockIdx.x*blockDim.x + threadIdx.x;
//     float items = size / (float) (blockDim.x * gridDim.x);
//     int batch = ceil(items);
//     if (index * batch >= size) {
//         return;
//     }
//     /*
//     to do the forward pass, we need to take the dot product of the current activations and the previous activations. This is gonna take some effort, so maybe we should
//     */
//     /*Step 1: Subtract current predictions from the actual output (same step as in logistic regression)
//     But there's a caveat involved in where we actually store the results*/
//     int currentLayer = model->nLayers-1;
//     float* current = activations+offsets[currentLayer]+(index*nClasses*batch);
//     float* out = outputs+(index*nClasses*batch);
//     float* deltaPtr = deltas+offsets[currentLayer]+(index*nClasses*batch);
//     int batch_size = min(size-(index*batch), batch);
//     //compute deltas for the last layer;
//     matrixSubtract(current, out, batch_size, model->layer_size[currentLayer+1], batch_size, model->layer_size[currentLayer+1], deltaPtr); //[10X10 vector]
//     int bias_index = model->layer_size[currentLayer+1]*index;
//     // printf("Bias index %d\n", bias_index);
//     for(int j = 0; j < model->layer_size[currentLayer+1]; j++) {
//         model->grad_biases[currentLayer][bias_index+j] = 0.0;
//         for(int i = 0; i < batch_size; i++) {
//             // printf("Delta ptr %d %d %f\n", index, i, deltaPtr[i*model->layer_size[currentLayer+1]+j]);
//             model->grad_biases[currentLayer][bias_index+j] += deltaPtr[i*model->layer_size[currentLayer+1]+j];
//         }
//     }
//     // //main loop
//     for(int i = currentLayer; i > 0; i--) {
//         dotProductTranspose(deltaPtr, model->weights[i], deltas+offsets[i-1]+(index*model->layer_size[i]*batch), batch_size, model->layer_size[i+1], model->layer_size[i], model->layer_size[i+1]);
//         deltaPtr = deltas+offsets[i-1]+(index*model->layer_size[i]*batch);
//         current = activations+offsets[i-1]+((index*model->layer_size[i]*batch));

//         //mulitply delta by the derivative of the sigmoid function
//         sigmoidD(current, batch_size, model->layer_size[i], deltaPtr);

//         //compute gradients with respect to the biases
//         bias_index = model->layer_size[i]*index;
//         for(int j = 0; j < model->layer_size[i]; j++) {
//             model->grad_biases[i-1][bias_index+j] = 0.0;
//             for(int k = 0; k < batch_size; k++) {
//                 model->grad_biases[i-1][bias_index+j] += deltaPtr[k*(model->layer_size[i])+j];
//             }
//         }
//     }
//     currentLayer = model->nLayers-1;
//     float * activationPtr; 
//     for(int i = currentLayer; i > 0; i--) {
//         deltaPtr = deltas+offsets[i]+(index*model->layer_size[i+1]*batch);
//         activationPtr = activations+offsets[i-1]+((index*model->layer_size[i]*batch)); 
//         int gradientIndex = (index*model->layer_size[i+1]*model->layer_size[i]);
//         dotProductTranspose(activationPtr, deltaPtr, model->gradients[i]+gradientIndex, batch_size, model->layer_size[i], batch_size, model->layer_size[i+1]);
//     }
//     deltaPtr = deltas+(index*model->layer_size[1]*batch);
//     int gradientIndex = (index*model->layer_size[0]*model->layer_size[1]);
//     dotProductTranspose(inputs+(model->layer_size[0]*batch*index), deltaPtr, model->gradients[0]+gradientIndex, batch_size, model->layer_size[0], batch_size, model->layer_size[1]);
// }

__global__ void auditDeltas(NeuralNetwork* model, float * deltas, int* offsets, int batches, int batch_size) {
    float* deltaPtr;
    for(int i = 1; i <= model->nLayers; i++) {
        printf("Deltas for layer %d %d\n", i, offsets[i-1]);
        deltaPtr = deltas+offsets[i-1];
        for(int j = 0; j < batch_size; j++) {
            for(int k = 0; k < model->layer_size[i]-1; k++) {
                printf("%f\t", deltaPtr[j*model->layer_size[i]+k]);
            }
            printf("%f\n", deltaPtr[(j+1)*(model->layer_size[i])-1]);
        }
    }
}
__global__ void auditGradients(NeuralNetwork* model) {
    printf("Audit\n");
    for(int i = 0; i < model->nLayers; i++) {
        printf("Gradients for weights %d\n", i);
        for(int j = 0; j < model->layer_size[i]; j++) {
            for(int k = 0; k < model->layer_size[i+1]-1; k++) {
                printf("%f\t", model->gradients[i][(j*model->layer_size[i+1])+k]);
            }
            printf("%f\n", model->gradients[i][(j+1)*model->layer_size[i+1]-1]);
        }

        // printf("Gradients for biases %d\n", i);
        // for(int j = 0; j < model->layer_size[i+1]-1; j++) {
        //     printf("%f\t", model->grad_biases[i][j]);
        // }
        // printf("%f\n", model->grad_biases[i][model->layer_size[i+1]-1]);
    }
    printf("Success\n");
}
__global__ void auditWeights(NeuralNetwork* model) {
    for(int i = 0; i < model->nLayers; i++) {
        printf("Weights at layer %d\n", i);
        for(int j = 0; j < model->layer_size[i]; j++) {
            for(int k = 0; k < model->layer_size[i+1]-1; k++) {
                printf("%f\t", model->weights[i][j*model->layer_size[i+1]+k]);
            }
            printf("%f\n", model->weights[i][(j*model->layer_size[i+1])+model->layer_size[i+1]-1]);
        }
    }
}

__global__ void test_func(float* mat, float* weights, float* prod, int vector_h, int vector_w) {
    printf("Successful significant\n");
    printf("%f %f\n", mat[0], mat[1]);
}

__global__ void ringReduce(NeuralNetwork* model, const int total_steps) {
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    for(int i = 0; i < model->nLayers; i++) {

        //reduce gradients[i]
        int step_size = (model->layer_size[i] * model->layer_size[i+1]);
        float step = (step_size) /(float) (blockDim.x * gridDim.x);
        int batch = ceil(step);
        // printf("Batch size for index %d of gradients %d: %d\n", index, i, batch);
        int start = index*batch;
        if(start >= step_size) {
            return;
        }
        for(int j = 1; j < total_steps; j++) {
            for(int k = start; k < min(start+batch, step_size); k++) {
                model->gradients[i][k] += model->gradients[i][k+(j*step_size)];
            }
        }

        //reduce biases[i]
        step_size = model->layer_size[i+1];
        step = step_size / (float) (blockDim.x * gridDim.x);
        batch = ceil(step);
        start = index*batch;
        if(start >= step_size) {
            return;
        }
        for(int j = 1; j < total_steps; j++) {
            for(int k = start; k < min(start+batch, step_size); k++) {
                // printf("Entry %d %d\n", j, k);
                model->grad_biases[i][k] += model->grad_biases[i][k+(j*step_size)];
            }
        }
    }
}

__global__ void backward_pass(NeuralNetwork* model, int batch_size, float learning_rate) {
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    //BATCH_SIZE*n_classes length vector
    for(int k = 0; k < model->nLayers; k++) {
        // printf("Layer %d\n", k);
        int size = (model->layer_size[k] * model->layer_size[k+1]);
        float step = size / (float) (blockDim.x * gridDim.x);
        int batch = ceil(step);
        int start = index*batch;
        // printf("Gradients for")
        // printf("Starting index %d %d %d\n", k, index, start);
        if(start >= size) {
            return;
        }
        for(int i = start; i < min(start+batch, size); i++) {
            (*model).gradients[k][i] *= (learning_rate) / (float) batch_size;
            (*model).weights[k][i] -=  (*model).gradients[k][i];
            // printf("WEIGHT AT %d %d: %f\n", k, i, (*model).weights[k][i]);
        }

        size = model->layer_size[k+1];
        step = size / (float) (blockDim.x * gridDim.x);
        batch = ceil(step);
        start = index*batch;
        if(start >= size) {
            return;
        }
        for(int i = start; i < min(start+batch, size); i++) {
            // printf("Entry %d %d\n", j, k);
            model->grad_biases[k][i] *= (learning_rate) / (float) batch_size;
            model->biases[k][i] -= model->grad_biases[k][i];
        }
    }
    // printf("Finish backward\n");
}

__global__ void dotProductSegmented(float* inputs, float* weights, float * product, int vector_h, int vector_w, int weight_h, int weight_w) {
    printf("This is called\n");
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    int index_x = blockIdx.z*blockDim.y + blockIdx.y;
    int index_y = threadIdx.z*gridDim.y + threadIdx.y;
    if(vector_h % (gridDim.y*gridDim.z) != 0 || weight_w % (blockDim.y*blockDim.z)) {
        printf("BAD ARGUMENTS\n");
        return;
    }
    int size_x = vector_h / (gridDim.y*gridDim.z);
    int size_y = weight_w / (blockDim.y*blockDim.z);
    printf("Size_x: %d\nSize_y: %d\n", size_x, size_y);
    float* out = product+(size_x*index_x*weight_w)+(size_y*index_y); 
    float* input = inputs+((size_x*index_x*vector_w));
    float* weight = weights+(size_y*index_y);
    for(int i = 0; i < size_x; i++) { //for every row in the first matrix
        for(int j = 0; j < size_y; j++) { //for every column in the second matrix
            out[i*weight_w+j] = 0.0;
            for(int k = 0; k < vector_w; k++) { //we compute the kth entry in row i of the INPUTS times the kth entry in column j of the WEIGHTS
                out[i*weight_w+j] += input[i*vector_w+k] * weight[k*weight_w+j];
                // printf("This %d %d %f %f\n", i, j, inputs[i*vector_w+k], weights[k*weight_w+j]);
            }
        }
    }
}

__global__ void dotProductSegmented(float* inputs, float* weights, float * product, int vector_h, int vector_w, int weight_h, int weight_w, float* bias) {
    printf("Successful call\n");
    printf("Input start %f %f\n", inputs[0], product[0]);
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    //we subdivide vector_h in "mini-batches" of size batch_size
    //we need to check that (blockDim.x * gridDim.x) is divisible by vector_h
    if(vector_h % (blockDim.x * gridDim.x) != 0) {
        printf("BAD ARGUMENT for global batch size\n");
        return;
    }
    int batch_size = vector_h / (blockDim.x * gridDim.x);
    int index_x = blockIdx.z*blockDim.y + blockIdx.y;
    int index_y = threadIdx.z*gridDim.y + threadIdx.y;
    //and we subdivide batch_size further by dividing by gridDim.y*gridDim.z
    if((batch_size % (gridDim.y*gridDim.z) != 0) || (weight_w % (blockDim.y*blockDim.z) != 0)) {
        printf("BAD ARGUMENTS\n");
        printf("Size_x %d %d\nSize_y %d %d\n", vector_h, gridDim.y*gridDim.z, weight_w, blockDim.y*blockDim.z);
        return;
    }
    if((batch_size < gridDim.y*gridDim.z) || (weight_w < blockDim.y*blockDim.z)) {
        printf("BAD ARGUMENTS\n");
        printf("Size_x %d %d\nSize_y %d %d\n", batch_size, gridDim.y*gridDim.z, weight_w, blockDim.y*blockDim.z);
        return; 
    }
    int size_x = batch_size / (gridDim.y*gridDim.z);
    int size_y = weight_w / (blockDim.y*blockDim.z);
    // printf("Start for %d %d %d\n", index, index*batch_size*vector_w, index*batch_size*weight_w);
    float* out = product+(index*batch_size*weight_w)+(size_x*index_x*weight_w)+(size_y*index_y); 
    float* input = inputs+(index*batch_size*vector_w)+((size_x*index_x*vector_w));
    float* weight = weights+(size_y*index_y);
    for(int i = 0; i < size_x; i++) { //for every row in the first matrix
        for(int j = 0; j < size_y; j++) { //for every column in the second matrix
            out[i*weight_w+j] = 0.0;
            for(int k = 0; k < vector_w; k++) { //we compute the kth entry in row i of the INPUTS times the kth entry in column j of the WEIGHTS
                out[i*weight_w+j] += input[i*vector_w+k] * weight[k*weight_w+j];
            }
            out[i*weight_w+j] += bias[j+(index_y*size_y)];
        }
    }
}

__global__ void sigmoidSegmented(float* inputs, int inputSize) {
    if(inputSize % (blockDim.x * gridDim.x) != 0) {
        printf("BAD ARGUMENT\n");
        return;
    }
    int blockSize = inputSize / (blockDim.x * gridDim.x);
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    sigmoid(inputs+(blockSize*index), blockSize);

}

__global__ void softmaxSegmented(float* product, int product_height, int product_width) {
    if(product_height % (blockDim.x * gridDim.x) != 0) {
        printf("BAD ARGUMENT\n");
        return;
    }
    int blockSize = product_height / (blockDim.x * gridDim.x);
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    softmax(product+(blockSize*index), blockSize, product_width);
}

