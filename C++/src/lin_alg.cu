#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include "../include/lin_alg.h"

//////////DEVICES////////

__device__ void softmax(float* product, int product_height, int product_width) {
    float total = 0.0;
    float logSumTotal = 0.0;
    for (int i = 0; i < product_height; i++) {
        total = 0.0;
        for (int j = 0; j < product_width; j++) {
            total += exp(product[i*product_width+j]);
        }
        logSumTotal = log(total);
        float prob_sums = 0.0;
        for (int j = 0; j < product_width; j++) {
            product[i*product_width+j] = exp(product[i*product_width+j] - logSumTotal);
            prob_sums += product[i*product_width+j];
        }

    }
}
__device__ void dotProduct(float* inputs, float* weights, float * product, int vector_h, int vector_w, int weight_h, int weight_w) {
    //initialize the matrix
    //dot product is ALWAYS computed as the rows of the first matrix by the columns of the second matrix
    for(int i = 0; i < vector_h; i++) { //for every row in the first matrix
        for(int j = 0; j < weight_w; j++) { //for every column in the second matrix
            product[i*weight_w+j] = 0.0;
            for(int k = 0; k < vector_w; k++) { //we compute the kth entry in row i of the INPUTS times the kth entry in column j of the WEIGHTS
                product[i*weight_w+j] += inputs[i*vector_w+k] * weights[k*weight_w+j];
            }
            // printf("Temp product: %d %d %f\n", i, j, product[i*weight_w+j]);
            //printf("%f\n", product[i][j]);
        }
    }
}

__device__ void dotProductTranspose(float* inputs, float* weights, float * product, int vector_h, int vector_w, int weight_h, int weight_w) {
    //remember that we want the resulting matrix to be of shape [vector_h, weight_w]
    for(int i = 0; i < vector_w; i++) {
        for(int j = 0; j < weight_w; j++) {
            product[i*weight_w+j] = 0.0;
            for(int k = 0; k < vector_h; k++) {
                product[i*weight_w+j] += inputs[k*vector_w+i] * weights[k*weight_w+j];
            }
        }
    }
}

__device__ void matrixSubtract(float * matrix1, float *matrix2, int m1_h, int m1_w, int m2_h, int m2_w, float* outVec) {
    if (m1_h == m2_h && m1_w == m2_w) {
        for (int i = 0; i < m1_h; i++) {
            for (int j = 0; j < m1_w; j++) {
                outVec[(i*m1_w)+j] = matrix1[(i*m1_w)+j]-matrix2[(i*m1_w)+j];
            }
        }
    }
}

__device__ void matrixAdd(float * matrix1, float * matrix2, int m1_h, int m1_w) {
    for(int i = 0; i < m1_h; i++) {
        for(int j = 0; j < m1_w; j++) {
            matrix1[i*m1_w+j] += matrix2[i*m1_w+j];
        }
    }
}

__device__ void matrixMultiplyByScalar(float* mat, int m1_h, int m1_w, float scalar) {
    for(int i = 0; i < m1_h; i++) {
        for(int j = 0; j < m1_w; j++) {
            mat[(i*m1_w)+j]*= scalar;
        }
    }
}
//////////GLOBALS////////
/*
*gradients: the gradient vector
begin_part: the index of where this threads cumulative gradients begin in the subsection
end_part: the index of where this threads cumulative gradients end in the subsection
total_steps: the total number of steps that each thread needs to perform in order to acheive the full cumulative gradient
step_size: the total distance in grads that we need to jump every time
*/
__global__ void ringReduce(float* gradients, const int total_steps, const int step_size, const int chunk_size) {
    //we achieve our reduction in two loops: update and set
    //in the update loop, we're simply calculating the cumulative sum of each part of the respective gradients
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    int begin_part = index*chunk_size;
    int end_part = (index+1)*chunk_size;
    for(int i = 1; i < total_steps; i++) {
        for(int j = begin_part; j < end_part; j++) {
            gradients[j] += gradients[(i*step_size)+j];
        }
    }
}

__global__ void ringReduce(LogisticRegression * model, const int total_steps, const int step_size, const int chunk_size) {
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    int begin_part = index*chunk_size;
    int end_part = (index+1)*chunk_size;
    for(int i = 1; i < total_steps; i++) {
        for(int j = begin_part; j < end_part; j++) {
            model->gradients[j] += model->gradients[(i*step_size)+j];
        }
    }
    // printf("Ring reduce\n");
}
__global__ void predict(float * inputs, float* weights, float * product, int size, int n_features, int n_classes) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int batch = size / (blockDim.x * gridDim.x);
    dotProduct(inputs+(i*n_features*batch), weights, product+(i*n_classes*batch), batch, n_features, n_features, n_classes);
    softmax(product+(i*n_classes*batch), batch, n_classes);
}

__global__ void predict(LogisticRegression* model, float* inputs, float* product, int size) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int batch = size / (blockDim.x * gridDim.x);
    dotProduct(inputs+(i*(model->nFeatures)*batch), model->weights, product+(i*(model->nClasses)*batch), batch, model->nFeatures, model->nFeatures, model->nClasses);
    softmax(product+(i*(model->nClasses)*batch), batch, (model->nClasses));
}
__global__ void forward_pass(float* inputs, float* weights, float* outputs, float* product, float* gradients, int size, int n_features, int n_classes) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int batch = size / (blockDim.x * gridDim.x);
    matrixSubtract(product+(i*n_classes*batch), outputs+(i*n_classes*batch), batch, n_classes, batch, n_classes, product+(i*n_classes*batch));
    dotProductTranspose(inputs+(i*batch*n_features), product+(i*batch*n_classes), gradients+(i*n_features*n_classes), batch, n_features, batch, n_classes);
}

__global__ void forward_pass(LogisticRegression* model, float* inputs, float* outputs, float* product, int size, int nClasses) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int batch = size / (blockDim.x * gridDim.x);
    matrixSubtract(product+(i*nClasses*batch), outputs+(i*nClasses*batch), batch, nClasses, batch, nClasses, product+(i*nClasses*batch));
    dotProductTranspose(inputs+(i*batch*(model->nFeatures)), product+(i*batch*(model->nClasses)), ((*model).gradients)+(i*(model->nClasses)*(model->nFeatures)), batch, (model->nFeatures), batch, (model->nClasses));
}

__global__ void backward_pass(float* weights, float * gradients, int batch_size, float learning_rate, int n_features, int n_classes) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    //BATCH_SIZE*n_classes length vector
    int batch = n_features / (blockDim.x * gridDim.x);
    matrixMultiplyByScalar(gradients+(i*batch*n_classes), batch, n_classes, learning_rate/(float) batch_size);
    matrixSubtract(weights+(i*n_classes*batch), gradients+(i*n_classes*batch), batch, n_classes, batch, n_classes, weights+(i*n_classes*batch));
}

__global__ void backward_pass(LogisticRegression* model, int batch_size, float learning_rate) {
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    //BATCH_SIZE*n_classes length vector
    int batch = model->nFeatures / (blockDim.x * gridDim.x);
    int start = index*batch*(model->nClasses);
    for(int i = 0; i < batch; i++) {
        for(int j = 0; j < model->nClasses; j++) {
            (*model).gradients[start+i*(model->nClasses)+j] *= (learning_rate / batch_size);
            (*model).weights[start+i*(model->nClasses)+j] -=  (*model).gradients[start+i*(model->nClasses)+j];
        }
    }
    // printf("Finish backward\n");
}

///NEURAL NETWORK CODE

__global__ void predict(NeuralNetwork* model, float* inputs, float* product, int size, int* offsets) {
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    int batch = size / (blockDim.x * gridDim.x);
    /*
    Each thread takes a chunk of the input data and feeds it all the way through the neural network, storing the intermediary results in product along the way.
    Just going to use softmax as the default activation as I'm more familiar with how that works anyways
    */
    dotProduct(inputs+(index*(model->layer_size[0])*batch), model->weights[0], product+index*(model->layer_size[1])*batch, batch, model->layer_size[0], model->layer_size[0], model->layer_size[1]);
    softmax(product+index*(model->layer_size[1])*batch, batch, (model->layer_size[1]));
    // printf("Current activation index: %d\n", activation_index);
    int layer = 0;
    printf("%f\n", product[0]);
    for(int j = 1; j < model->nLayers; j++) {
        printf("Exec %d\n", offsets[j-1]);
        dotProduct(product+offsets[j-1]+(index*(model->layer_size[j])*batch), model->weights[j], product+offsets[j]+(index*(model->layer_size[j+1])*batch), batch, model->layer_size[j], model->layer_size[j], model->layer_size[j+1]);
        softmax(product+offsets[j]+(index*(model->layer_size[j+1])*batch), batch, model->layer_size[j+1]);
    }
    printf("Previously %d %f\n", offsets[model->nLayers-1], *(product+offsets[model->nLayers-1]));
    printf("End of predict\n");
}

__global__ void forward_pass(NeuralNetwork* model, float* inputs, float* outputs, float* activations, int * offsets, int size, int nClasses) {
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    int batch = size / (blockDim.x * gridDim.x);
    /*
    to do the forward pass, we need to take the dot product of the current activations and the previous activations. This is gonna take some effort, so maybe we should
    */
    /*Step 1: Subtract current predictions from the actual output (same step as in logistic regression)
    But there's a caveat involved in where we actually store the results*/
    int currentLayer = model->nLayers-1;
    // printf("Activation index: %d\n", activation_index + (index*batch));
    float* current = activations+offsets[currentLayer];
    float* prev = activations+offsets[currentLayer-1];
    matrixSubtract(current+(index*nClasses*batch), outputs+(index*nClasses*batch), batch, nClasses, batch, nClasses, current+(index*nClasses*batch));
    dotProductTranspose(prev+(index*batch*(model->layer_size[currentLayer])), current+(index*nClasses*batch), ((*model).gradients[currentLayer])+(index*(nClasses)*(model->layer_size[currentLayer])), batch, (model->layer_size[currentLayer]), batch, (nClasses));
    // printMatrix(model->gradients[currentLayer-1]+index*(nClasses)*(model->layer_size[currentLayer]),);
    currentLayer--;
    // current = activations[currentLayer];
    // prev = activations[currentLayer-1];
    // for(int i = currentLayer; i > 0; i--) {
    //     //do something else
    //     printf("Don't call this\n");
    // }
    // matrixSubtract(inputs+(index*model->layer_size[0]*batch), current+(index*batch*model->layer_size[0]), batch, model->layer_size[0], batch, model->layer_size[0], current+(index*batch*model->layer_size[0]));
    // dotProductTranspose(activations+(index*model->layer_size[0]*batch), activations+offsets[i]+(index*model->layer_size[1]*batch), ((*model).gradients[0])+(index*(model->layer_size[0])*(model->layer_size[1])), batch, (model->layer_size[0]), batch, (model->layer_size[1]));
    // printf("Success of forward pass\n");
}

__global__ void ringReduce(NeuralNetwork* model, const int total_steps) {
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    for(int i = 0; i < model->nLayers; i++) {
        int step_size = (model->layer_size[i] * model->layer_size[i+1]);
        int batch = (model->layer_size[i] * model->layer_size[i+1]) /(blockDim.x * gridDim.x);
        int start = index*batch;
        int end = (index+1)*batch;

        for(int j = 1; j < total_steps; j++) {
            for(int k = start; k < end; k++) {
                // printf("Entry %d %d\n", j, k);
                model->gradients[i][k] += model->gradients[i][k+(j*step_size)];
            }
        }
    }
}

__global__ void backward_pass(NeuralNetwork* model, int batch_size, float learning_rate) {
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    //BATCH_SIZE*n_classes length vector
    for(int k = 0; k < model->nLayers; k++) {
        // printf("Layer %d\n", k);
        int batch = (model->layer_size[k] * model->layer_size[k+1]) /(blockDim.x * gridDim.x);
        int start = index*batch;
        for(int i = 0; i < batch; i++) {
            printf("Gradient at %d %d %f\n", k, i, (*model).gradients[k][start+i]);
            (*model).gradients[k][start+i] *= (learning_rate / batch_size);
            (*model).weights[k][start+i] -=  (*model).gradients[k][start+i];
        }
    }
    // printf("Finish backward\n");
}
