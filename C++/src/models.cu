#include "hip/hip_runtime.h"
#include "../include/utils.h"
#include "../include/lin_alg.h"
#include "../include/models.h"
#include <chrono> 
#include <iostream>

float* transferMatrixToDevice(float *matrix, int height, int width) {
    float* deviceMatrix;
    hipMalloc(&deviceMatrix, height*width*sizeof(float));
    for(int i = 0; i < height; i++) {
        hipMemcpy(deviceMatrix+(i*width), matrix+(i*width), sizeof(float)*width, hipMemcpyHostToDevice);
    }
    return deviceMatrix;
}

// LogisticRegression * copyModelToHost(LogisticRegression *model, LogisticRegression *start) {
//     LogisticRegression* host;
//     host->weights = initializeFlatRandomArray(nFeatures, numClasses);
//     host->gradients = (float*)malloc(nFeatures*numClasses*sizeof(float));
// }
LogisticRegression* copyModelToGPU(LogisticRegression *model, int nWorkers, int nThreadsPerWorker) {
    //define pointer for GPU's copy of the model
    LogisticRegression* d_model;
    //allocate space in the GPU memory for the model
    hipMalloc(&d_model, sizeof(LogisticRegression));

    //declare all of the pointers that we need on the GPU (weights and gradients) and pass them to device
    float *d_weights;
    hipMalloc(&d_weights, model->nFeatures*model->nClasses*sizeof(float));
    hipMemcpy(d_weights, (*model).weights, model->nFeatures*model->nClasses*sizeof(float), hipMemcpyHostToDevice);
    float *d_gradients;
    hipMalloc(&d_gradients, nThreadsPerWorker*nWorkers*model->nFeatures*model->nClasses*sizeof(float));
    hipMemcpy(d_gradients, (*model).gradients, nThreadsPerWorker*nWorkers*model->nFeatures*model->nClasses*sizeof(float), hipMemcpyHostToDevice);
    //create temp model
    LogisticRegression temp = *model;
    temp.weights = d_weights;
    temp.gradients = d_gradients;
    temp.nFeatures = model->nFeatures;
    temp.nClasses = model->nClasses;
    //pass temp model to GPU
    hipMemcpy(d_model, &temp, sizeof(LogisticRegression), hipMemcpyHostToDevice);
    return d_model;
}

NeuralNetwork * copyModelToGPU(NeuralNetwork *model, int nWorkers, int nThreadsPerWorker) {
    NeuralNetwork* d_model;
    int * nLayers;
    float **d_weights;
    float **d_weights_t;
    float **d_biases;
    float **d_gradients;
    float **d_grad_biases;
    //allocate all of the memory that we need to CUDA
    hipMalloc(&d_model, sizeof(NeuralNetwork));
    hipMalloc(&nLayers, (model->nLayers+1)*sizeof(int));
    hipMemcpy(nLayers, model->layer_size, (model->nLayers+1)*sizeof(int), hipMemcpyHostToDevice);
    // // hipMalloc(&d_weights, (model->nLayers)*sizeof(float*));
    // hipMalloc(&d_biases, (model->nLayers)*sizeof(float*));
    float **temp_weights = new float*[model->nLayers];
    float **temp_weights_t = new float*[model->nLayers];
    float **temp_biases = new float*[model->nLayers];
    float **temp_gradients = new float*[model->nLayers];
    float **temp_grad_biases = new float*[model->nLayers];
    for(int i = 1; i < model->nLayers+1; i++) {
        hipMalloc(&temp_weights[i-1], model->layer_size[i-1]*model->layer_size[i]*sizeof(float));
        hipMemcpy(temp_weights[i-1], model->weights[i-1], model->layer_size[i-1]*model->layer_size[i]*sizeof(float), hipMemcpyHostToDevice);
        hipMalloc(&temp_biases[i-1], model->layer_size[i]*sizeof(float));
        hipMemcpy(temp_biases[i-1], model->biases[i-1], model->layer_size[i]*sizeof(float), hipMemcpyHostToDevice);
        hipMalloc(&temp_gradients[i-1], nThreadsPerWorker*nWorkers*model->layer_size[i-1]*model->layer_size[i]*sizeof(float));
        hipMalloc(&temp_weights_t[i-1], model->layer_size[i-1]*model->layer_size[i]*sizeof(float));
        // hipMemcpy(temp_gradients[i-1], model->gradients[i-1], nThreadsPerWorker*nWorkers*model->layer_size[i-1]*model->layer_size[i]*sizeof(float), hipMemcpyHostToDevice);
        hipMalloc(&temp_grad_biases[i-1],  nThreadsPerWorker*nWorkers*model->layer_size[i]*sizeof(float));
    }
    hipMalloc(&d_gradients, (model->nLayers)*sizeof(float*));
    hipMemcpy(d_gradients, temp_gradients, (model->nLayers)*sizeof(float*), hipMemcpyHostToDevice);
    hipMalloc(&d_grad_biases, (model->nLayers)*sizeof(float*));
    hipMemcpy(d_grad_biases, temp_grad_biases, (model->nLayers)*sizeof(float*), hipMemcpyHostToDevice);
    hipMalloc(&d_biases, (model->nLayers)*sizeof(float*));
    hipMemcpy(d_biases, temp_biases, (model->nLayers)*sizeof(float*), hipMemcpyHostToDevice);
    hipMalloc(&d_weights, (model->nLayers)*sizeof(float*));
    hipMemcpy(d_weights, temp_weights, (model->nLayers)*sizeof(float*), hipMemcpyHostToDevice);
    hipMalloc(&d_weights_t, (model->nLayers)*sizeof(float*));
    hipMemcpy(d_weights_t, temp_weights_t, (model->nLayers)*sizeof(float*), hipMemcpyHostToDevice);
    NeuralNetwork temp = *model;
    temp.nClasses = model->nClasses;
    temp.nLayers = model->nLayers;
    temp.layer_size = nLayers;
    temp.weights = d_weights;
    temp.weight_transpose = d_weights_t;
    temp.gradients = d_gradients;
    temp.biases = d_biases;
    temp.grad_biases = d_grad_biases;
    temp.lambda = model->lambda;
    hipMemcpy(d_model, &temp, sizeof(NeuralNetwork), hipMemcpyHostToDevice);
    return d_model;
}

void train(LogisticRegression *model, float* train_input, std::vector<std::vector<int>>& train_labels, float* test_input, std::vector<std::vector<int>>& test_labels, 
    int nEpochs, int batch_size, int total_size, int test_size, float learning_rate, int nWorkers, int nThreadsPerWorker) {
    //since we can't directly access device variables from the host function, we'll have to do everything here
    LogisticRegression *d_model = copyModelToGPU(model, nWorkers, nThreadsPerWorker);
    std::cout << "TEST SIZE " << test_size << std::endl;
    //copy train data
    float *d_inputs;
    //copy weights
    hipMalloc(&d_inputs, total_size*(model->nFeatures)*sizeof(float));
    hipMemcpy(d_inputs, train_input, total_size*(model->nFeatures)*sizeof(float), hipMemcpyHostToDevice);

    //copy test data
    float *d_test_inputs;
    hipMalloc(&d_test_inputs, test_size*(model->nFeatures)*sizeof(float));
    hipMemcpy(d_test_inputs, test_input, test_size*(model->nFeatures)*sizeof(float), hipMemcpyHostToDevice);

    //convert labels to one hot encoding
    float * one_hot = (float *)malloc(sizeof(float) * total_size * model->nClasses);
    for (int i = 0; i < total_size; i++) {
        for(int j = 0; j < model->nClasses; j++) {
            one_hot[i*model->nClasses+j] = 0;
        }
        one_hot[i*(model->nClasses)+train_labels[i][0]] = 1.0;
    }
    //pass labels to GPU
    float *d_outputs = transferMatrixToDevice(one_hot, total_size, model->nClasses);

    //initialize array for storing predictions on host
    float * predictions = (float*)malloc(batch_size*(model->nClasses)*sizeof(float));
    float * d_product = transferMatrixToDevice(predictions, batch_size, model->nClasses);
    //initialize array for storing predictions of test set on host
    float * test_predictions = (float*)malloc(test_size*model->nClasses*sizeof(float));
    float * d_test_product = transferMatrixToDevice(test_predictions, test_size, model->nClasses);
    //define metrics
    int correct = 0;
    double logLoss = 0.0;
    float accuracy = 0.0;
    for(int i = 0; i < nEpochs; i++) {
        correct = 0;
        logLoss = 0;
        accuracy = 0.0;

        for(int j = 0; j < total_size; j+=batch_size) {
            predict<<<nWorkers, nThreadsPerWorker>>>(d_model, d_inputs+(j*model->nFeatures), d_product, batch_size);
            // predict<<<nWorkers, nThreadsPerWorker>>>(d_inputs+(j*model->nFeatures), d_model->weights, d_product, batch_size, model->nFeatures, model->nClasses);
            hipDeviceSynchronize();
            hipMemcpy(predictions, d_product, batch_size*(model->nClasses)*sizeof(float), hipMemcpyDeviceToHost);
            // printf("Probabilities\n");
            // printMatrix(product, batch_size, model->nClasses);
            // correct += getAccuracy(predictions, train_labels, batch_size, model->nClasses, j);
            // logLoss += crossEntropyLoss(predictions, train_labels, batch_size, model->nClasses, j);
            forward_pass<<<nWorkers, nThreadsPerWorker>>>(d_model, d_inputs+(j*(model->nFeatures)), d_outputs+(j*(model->nClasses)), d_product, batch_size, model->nClasses);
            hipDeviceSynchronize();

            // auto endForward = std::chrono::system_clock::now();
            // std::chrono::duration<double> elapsed_forward = endForward - initForward;
            // std::cout << "Finished forward pass in " << elapsed_forward.count() << " seconds" << std::endl;
            // //backward pass
            // std::cout << "Starting backward pass..." << std::endl;
            auto initBackward = std::chrono::system_clock::now();
            ringReduce<<<nWorkers, nThreadsPerWorker>>>(d_model, nThreadsPerWorker*nWorkers, model->nFeatures*model->nClasses, model->nFeatures*model->nClasses/(nThreadsPerWorker*nWorkers));
            hipDeviceSynchronize();
            // float * gradients = (float*)malloc(nFeatures*model->nClasses*sizeof(float));
            // printf("Weights\n");
            // printMatrix(model->weights, model->nFeatures, model->nClasses);
            backward_pass<<<nWorkers, nThreadsPerWorker>>>(d_model, batch_size, learning_rate);
            hipDeviceSynchronize();

        }
        accuracy = correct / (float) total_size;
        printf("Accuracy: %f%%\n", accuracy*100);
        printf("Log loss: %f\n", logLoss);
        predict<<<10, 10>>>(d_model, d_test_inputs, d_test_product, test_size);
        hipDeviceSynchronize();
        std::cout << "Finished eval" << std::endl;
        hipMemcpy(test_predictions, d_test_product, test_size*(model->nClasses)*sizeof(float), hipMemcpyDeviceToHost);
        int test_correct = getAccuracy(test_predictions, test_labels, test_size, model->nClasses, 0);
        double test_loss = crossEntropyLoss(test_predictions, test_labels, test_size, model->nClasses, 0);
        printf("Test log loss: %f\nTest accuracy %f%%\n", test_loss, test_correct / (float) test_size * 100);
    }
    hipFree(d_model);
    hipFree(d_inputs);
    hipFree(d_test_inputs);
    hipFree(d_outputs);
    hipFree(d_product);
    hipFree(d_test_product);
}

void train(NeuralNetwork *model, float* train_input, std::vector<std::vector<int>>& train_labels, float* test_input, std::vector<std::vector<int>>& test_labels, 
int nEpochs, int batch_size, int total_size, int test_size, float learning_rate, int nWorkers, int nThreadsPerWorker) {
    printf("Train network\n");
    NeuralNetwork *d_model = copyModelToGPU(model, nWorkers, nThreadsPerWorker);
    std::cout << "TEST SIZE " << test_size << std::endl;
    //copy train data
    float *d_inputs;
    //copy weights
    hipError_t error;
    error = hipMalloc(&d_inputs, total_size*(model->layer_size[0])*sizeof(float));
    if(error != hipSuccess) {
        std::cout << "Problem with copying" << std::endl;
    }
    error = hipMemcpy(d_inputs, train_input, total_size*(model->layer_size[0])*sizeof(float), hipMemcpyHostToDevice);
    if(error != hipSuccess) {
        std::cout << "Problem" << std::endl;
    }
    //copy test data
    float *d_test_inputs;
    hipMalloc(&d_test_inputs, test_size*(model->layer_size[0])*sizeof(float));
    hipMemcpy(d_test_inputs, test_input, test_size*(model->layer_size[0])*sizeof(float), hipMemcpyHostToDevice);

    //convert labels to one hot encoding
    float * one_hot = (float *)malloc(sizeof(float) * total_size * model->nClasses);
    for (int i = 0; i < total_size; i++) {
        for(int j = 0; j < model->nClasses; j++) {
            one_hot[i*model->nClasses+j] = 0;
        }
        one_hot[i*(model->nClasses)+train_labels[i][0]] = 1.0;
    }
    //pass labels to GPU
    float *d_outputs = transferMatrixToDevice(one_hot, total_size, model->nClasses);

    //initialize array for storing intermediary activation functions on GPU
    /*the super nice thing about the parallelized computation of neural networks is 
    ALL of the activation functions take the form of (BATCH_SIZE, layer_size)
    Which means we can likely have all of the activations stored via one pointer and only
    have to allocate the memory ONCE. However, since I have absolutely no idea what I'm doing,
    I'm gonna stay away from that for now.

    Since double pointers don't want to cooperate for some reason, and since it doesn't make sense
    for this huge block of memory to be allocated several different times randomly in memory, we allocate a single block
    of memory as well as an integer array to keep track of the offsets of each "block" in memory.
    */
    int activations_size = 0;
    int * offsets = new int[model->nLayers];
    for(int i = 1; i <= model->nLayers; i++) {
        offsets[i-1] = (batch_size * activations_size);
        activations_size += model->layer_size[i];
        printf("Offset at %d %d\n", i-1, offsets[i-1]);
    }
    float * d_activations = new float[batch_size*activations_size];
    float * activations = new float[batch_size*activations_size];
    printf("Activations size: %d\n", batch_size*activations_size);
    //device pointers
    int * d_offsets;
    hipMalloc(&d_activations, activations_size*batch_size*sizeof(float));
    hipMalloc(&d_offsets, model->nLayers*sizeof(int));
    for(int i = 0; i < activations_size*batch_size; i++) {
        activations[i] = 1;
    }
    hipMemcpy(d_activations, activations, activations_size*batch_size*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_offsets, offsets, model->nLayers*sizeof(int), hipMemcpyHostToDevice);

    //deltas
    float * d_deltas = new float[batch_size*activations_size];
    hipMalloc(&d_deltas, activations_size*batch_size*sizeof(float));
    hipMemcpy(d_activations, activations, activations_size*batch_size*sizeof(float), hipMemcpyHostToDevice);
    // float * d_product = transferMatrixToDevice(activations, batch_size, activations_size);
    // //initialize array for storing predictions of test set on host
    float * test_predictions = (float*)malloc(test_size*model->nClasses*sizeof(float));
    float * d_test_product = transferMatrixToDevice(test_predictions, test_size, model->nClasses);
    //define metrics
    int correct = 0;
    double logLoss = 0.0;
    float accuracy = 0.0;
    auto startTrain = std::chrono::system_clock::now();
    for(int i = 0; i < nEpochs; i++) {
        correct = 0;
        logLoss = 0;
        accuracy = 0.0;

        for(int j = 0; j < 1000; j+=batch_size) {
            //pass inputs through the network
            setTranspose<<<1,1>>>(d_model);
            hipDeviceSynchronize();
            auto startForward = std::chrono::system_clock::now();
            predict<<<nWorkers, nThreadsPerWorker>>>(d_model, d_inputs+(j*model->layer_size[0]), d_activations, d_offsets, batch_size);
            hipDeviceSynchronize();
            auto endForward = std::chrono::system_clock::now();
            std::chrono::duration<double> elapsed_forward = endForward - startForward;
            std::cout << "Finished forward pass in " << elapsed_forward.count() << " seconds" << std::endl;
            float* predictions = (float*)malloc(activations_size*batch_size*sizeof(float));
            error = hipMemcpy(predictions, d_activations, activations_size*batch_size*sizeof(float), hipMemcpyDeviceToHost);
            // for(int k = 0; k < model->nLayers; k++) {
            //     printf("Activations at layer %d\n", k);
            //     printMatrix(predictions+offsets[k], batch_size, model->layer_size[k+1]);
            // }
            correct += getAccuracy(predictions+offsets[1], train_labels, batch_size, model->nClasses, j);
            logLoss += crossEntropyLoss(predictions+offsets[1], train_labels, batch_size, model->nClasses, j);
            // printf("Accuracy: %f%%\n", correct / (float) batch_size * 100);
            // printf("Log loss %f\n", logLoss);
            // //compute gradients in forward_pass
            auto startBackward = std::chrono::system_clock::now();
            backprop<<<nWorkers, nThreadsPerWorker>>>(d_model, d_inputs+(j*(model->layer_size[0])), d_outputs+(j*(model->nClasses)), d_activations, d_deltas, d_offsets, batch_size, model->nClasses);
            hipDeviceSynchronize();
            auto endBackward = std::chrono::system_clock::now();
            std::chrono::duration<double> elapsed_backward = endBackward - startBackward;
            std::cout << "Finished backward pass in " << elapsed_backward.count() << " seconds" << std::endl;
            auto startReduce = std::chrono::system_clock::now();
            ringReduce<<<nWorkers, nThreadsPerWorker>>>(d_model, nWorkers*nThreadsPerWorker);
            hipDeviceSynchronize();
            auto endReduce = std::chrono::system_clock::now();
            std::chrono::duration<double> elapsed_reduce = endReduce - startReduce;
            std::cout << "Finished ring reduce in " << elapsed_reduce.count() << " seconds" << std::endl;
            // auditDeltas<<<1,1>>>(d_model, d_deltas, d_offsets, nWorkers*nThreadsPerWorker, batch_size);
            // hipDeviceSynchronize();
            // auditGradients<<<1,1>>>(d_model);
            // hipDeviceSynchronize();
            auto startUpdate = std::chrono::system_clock::now();
            backward_pass<<<nWorkers, nThreadsPerWorker>>>(d_model, batch_size, learning_rate);
            hipDeviceSynchronize();
            auto endUpdate = std::chrono::system_clock::now();
            std::chrono::duration<double> elapsed_update = endUpdate - startUpdate;
            std::cout << "Finished weight update in " << elapsed_update.count() << " seconds" << std::endl;
        }
        accuracy = correct / (float) total_size;
        printf("End of epoch %d\n", i+1);
        printf("Accuracy: %f%%\n", accuracy*100);
        printf("Log loss: %f\n", logLoss);
    }
    auto endTrain = std::chrono::system_clock::now();
    std::chrono::duration<double> elapsed_forward = endTrain - startTrain;
    std::cout << "Finished forward pass in " << elapsed_forward.count() << " seconds" << std::endl;
    hipFree(d_model);
    hipFree(d_inputs);
    hipFree(d_test_inputs);
    hipFree(d_outputs);
    hipFree(d_activations);
    hipFree(d_test_product);
    hipFree(d_deltas);
}