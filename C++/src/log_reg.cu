#include "hip/hip_runtime.h"
#include "../include/log_reg.h"
#include "../include/lin_alg.h"
#include "../include/utils.h"
#include <iostream>
#include <chrono> 

__global__ void ringReduce(LogisticRegression * model, const int total_steps, const int step_size, const int chunk_size) {
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    int begin_part = index*chunk_size;
    int end_part = (index+1)*chunk_size;
    for(int i = 1; i < total_steps; i++) {
        for(int j = begin_part; j < end_part; j++) {
            model->gradients[j] += model->gradients[(i*step_size)+j];
        }
    }
    // printf("Ring reduce\n");
}

// __global__ void predict(LogisticRegression* model, float* inputs, float* product, int size) {
//     int i = blockIdx.x*blockDim.x + threadIdx.x;
//     int batch = size / (blockDim.x * gridDim.x);
//     dotProduct(inputs+(i*(model->nFeatures)*batch), model->weights, product+(i*(model->nClasses)*batch), batch, model->nFeatures, model->nFeatures, model->nClasses);
//     softmax(product+(i*(model->nClasses)*batch), batch, (model->nClasses));
// }

// __global__ void forward_pass(LogisticRegression* model, float* inputs, float* outputs, float* product, int size, int nClasses) {
//     int i = blockIdx.x*blockDim.x + threadIdx.x;
//     int batch = size / (blockDim.x * gridDim.x);
//     matrixSubtract(product+(i*nClasses*batch), outputs+(i*nClasses*batch), batch, nClasses, batch, nClasses, product+(i*nClasses*batch));
//     dotProductTranspose(inputs+(i*batch*(model->nFeatures)), product+(i*batch*(model->nClasses)), ((*model).gradients)+(i*(model->nClasses)*(model->nFeatures)), batch, (model->nFeatures), batch, (model->nClasses));
// }

__global__ void backward_pass(LogisticRegression* model, int batch_size, float learning_rate) {
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    //BATCH_SIZE*n_classes length vector
    int batch = model->nFeatures / (blockDim.x * gridDim.x);
    int start = index*batch*(model->nClasses);
    for(int i = 0; i < batch; i++) {
        for(int j = 0; j < model->nClasses; j++) {
            (*model).gradients[start+i*(model->nClasses)+j] *= (learning_rate / batch_size);
            (*model).weights[start+i*(model->nClasses)+j] -=  (*model).gradients[start+i*(model->nClasses)+j];
        }
    }
    // printf("Finish backward\n");
}

LogisticRegression* copyModelToGPU(LogisticRegression *model, int nWorkers, int nThreadsPerWorker) {
    //define pointer for GPU's copy of the model
    LogisticRegression* d_model;
    //allocate space in the GPU memory for the model
    hipMalloc(&d_model, sizeof(LogisticRegression));

    //declare all of the pointers that we need on the GPU (weights and gradients) and pass them to device
    float *d_weights;
    hipMalloc(&d_weights, model->nFeatures*model->nClasses*sizeof(float));
    hipMemcpy(d_weights, (*model).weights, model->nFeatures*model->nClasses*sizeof(float), hipMemcpyHostToDevice);
    float *d_gradients;
    hipMalloc(&d_gradients, nThreadsPerWorker*nWorkers*model->nFeatures*model->nClasses*sizeof(float));
    hipMemcpy(d_gradients, (*model).gradients, nThreadsPerWorker*nWorkers*model->nFeatures*model->nClasses*sizeof(float), hipMemcpyHostToDevice);
    //create temp model
    LogisticRegression temp = *model;
    temp.weights = d_weights;
    temp.gradients = d_gradients;
    temp.nFeatures = model->nFeatures;
    temp.nClasses = model->nClasses;
    //pass temp model to GPU
    hipMemcpy(d_model, &temp, sizeof(LogisticRegression), hipMemcpyHostToDevice);
    return d_model;
}

void train(LogisticRegression *model, float* train_input, std::vector<std::vector<int>>& train_labels, float* test_input, std::vector<std::vector<int>>& test_labels, 
    int nEpochs, int batch_size, int total_size, int test_size, float learning_rate, int nWorkers, int nThreadsPerWorker) {
    //since we can't directly access device variables from the host function, we'll have to do everything here
    LogisticRegression *d_model = copyModelToGPU(model, nWorkers, nThreadsPerWorker);
    std::cout << "TEST SIZE " << test_size << std::endl;
    //copy train data
    float *d_inputs;
    //copy weights
    hipMalloc(&d_inputs, total_size*(model->nFeatures)*sizeof(float));
    hipMemcpy(d_inputs, train_input, total_size*(model->nFeatures)*sizeof(float), hipMemcpyHostToDevice);

    //copy test data
    float *d_test_inputs;
    hipMalloc(&d_test_inputs, test_size*(model->nFeatures)*sizeof(float));
    hipMemcpy(d_test_inputs, test_input, test_size*(model->nFeatures)*sizeof(float), hipMemcpyHostToDevice);

    //convert labels to one hot encoding
    float * one_hot = (float *)malloc(sizeof(float) * total_size * model->nClasses);
    for (int i = 0; i < total_size; i++) {
        for(int j = 0; j < model->nClasses; j++) {
            one_hot[i*model->nClasses+j] = 0;
        }
        one_hot[i*(model->nClasses)+train_labels[i][0]] = 1.0;
    }
    //pass labels to GPU
    float *d_outputs = transferMatrixToDevice(one_hot, total_size, model->nClasses);

    //initialize array for storing predictions on host
    float * predictions = (float*)malloc(batch_size*(model->nClasses)*sizeof(float));
    float * d_product = transferMatrixToDevice(predictions, batch_size, model->nClasses);
    //initialize array for storing predictions of test set on host
    float * test_predictions = (float*)malloc(test_size*model->nClasses*sizeof(float));
    float * d_test_product = transferMatrixToDevice(test_predictions, test_size, model->nClasses);
    //define metrics
    int correct = 0;
    double logLoss = 0.0;
    float accuracy = 0.0;
    for(int i = 0; i < nEpochs; i++) {
        correct = 0;
        logLoss = 0;
        accuracy = 0.0;

        for(int j = 0; j < total_size; j+=batch_size) {

            // auto endForward = std::chrono::system_clock::now();
            // std::chrono::duration<double> elapsed_forward = endForward - initForward;
            // std::cout << "Finished forward pass in " << elapsed_forward.count() << " seconds" << std::endl;
            // //backward pas

        }
        accuracy = correct / (float) total_size;
        printf("Accuracy: %f%%\n", accuracy*100);
        printf("Log loss: %f\n", logLoss);
        hipDeviceSynchronize();
        std::cout << "Finished eval" << std::endl;
        hipMemcpy(test_predictions, d_test_product, test_size*(model->nClasses)*sizeof(float), hipMemcpyDeviceToHost);
        int test_correct = getAccuracy(test_predictions, test_labels, test_size, model->nClasses, 0);
        double test_loss = crossEntropyLoss(test_predictions, test_labels, test_size, model->nClasses, 0);
        printf("Test log loss: %f\nTest accuracy %f%%\n", test_loss, test_correct / (float) test_size * 100);
    }
    hipFree(d_model);
    hipFree(d_inputs);
    hipFree(d_test_inputs);
    hipFree(d_outputs);
    hipFree(d_product);
    hipFree(d_test_product);
}
