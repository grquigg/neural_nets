#include <gtest/gtest.h>
#include "../include/lin_alg.h"
#include "../include/utils.h"
#include "../include/models.h"

TEST(SegmentedDotProduct, SingleThreaded) {
    int nWorkers = 1, nThreadsPerWorker = 1;
    float arr1[6] = {1,2,3,4,5,6};
    float arr2[12] = {-1,-2,-3,-4,-5,-6,-7,-8,-9,-10,-11,-12};
    float product[8];
    float correct_ans[8] = {-38.0f, -44.0f, -50.0f, -56.0f, -83.0f, -98.0f, -113.0f, -128.0f};
    float *darr1;
    float *darr2;
    float *dproduct;
    hipMalloc(&darr1, 6*sizeof(float));
    hipMalloc(&darr2, 12*sizeof(float));
    hipMalloc(&dproduct, 8*sizeof(float));
    hipMemcpy(darr1, arr1, 6*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(darr2, arr2, 12*sizeof(float), hipMemcpyHostToDevice);
    dim3 nBlocks(nWorkers, 1, 1);
    dim3 nThreads(nThreadsPerWorker, 1, 1);
    dotProductSegmented<<<nBlocks, nThreads>>>(darr1, darr2, dproduct, 2, 3, 3, 4);
    hipDeviceSynchronize();
    hipMemcpy(product, dproduct, 8*sizeof(float), hipMemcpyDeviceToHost);
    for(int i = 0; i < 2; i++) {
        for(int j = 0; j < 4; j++) {
            EXPECT_EQ(product[i*4+j], correct_ans[i*4+j]);
        }
    }
}

TEST(SegmentedDotProduct, DotProductSingleThreadedEx1) { //this is based on the Backprop example 1 from 589 HW4
  int nWorkers = 1;
  int nThreadsPerWorker = 1;
  int layers[2] = {1, 2};
  float correct[4] = {0.413, 0.326, 0.442, 0.384};
  float *weights[1];
  weights[0] = (float*)malloc(2*sizeof(float));
  weights[0][0] = 0.1f;
  weights[0][1] = 0.2f;
  float *biases[1];
  biases[0] = (float*)malloc(2*sizeof(float));
  biases[0][0] = 0.4f;
  biases[0][1] = 0.3f;
  NeuralNetwork* model = buildModel(1, layers, weights, biases, 1.0, 1, 1);
  float input[2] = {0.13f, 0.42f};
  float product[4] = {1.0f, 1.0f, 1.0f, 1.0f};
  float *d_weights;
  float *d_input;
  float *d_product;
  float *d_bias;
  hipMalloc(&d_bias, 2*sizeof(float));
  hipMemcpy(d_bias, model->biases[0], 2*sizeof(float), hipMemcpyHostToDevice);
  hipMalloc(&d_weights, 2*sizeof(float));
  hipMemcpy(d_weights, model->weights[0], 2*sizeof(float), hipMemcpyHostToDevice);
  hipMalloc(&d_input, 2*sizeof(float));
  hipMemcpy(d_input, input, 2*sizeof(float), hipMemcpyHostToDevice);
  hipMalloc(&d_product, 4*sizeof(float));
  dim3 nBlocks(nWorkers, 1, 1);
  dim3 nThreads(nThreadsPerWorker, 1, 1);
  dotProductSegmented<<<nBlocks, nThreads>>>(d_input, d_weights, d_product, 2, model->layer_size[0], model->layer_size[0], model->layer_size[1], d_bias);
  hipDeviceSynchronize();
  float *prod = (float*)malloc(4*sizeof(float));
  hipMemcpy(prod, d_product, 4*sizeof(float), hipMemcpyDeviceToHost);
  for(int i = 0; i < 4; i++) {
    EXPECT_FLOAT_EQ(prod[i], correct[i]);
  }
}

TEST(SegmentedDotProduct, DotProductSingleThreadedEx2) { //REMEMBER THAT WE'RE TAKING THE TRANSPOSE OF THE MATRIX IN THE EXAMPLE
  int nWorkers = 1;
  int nThreadsPerWorker = 1;
  int layers[2] = {2, 4};
  float correct[8] = {0.74f, 1.1192f, 0.3564f, 0.8744f, 0.55250f, 0.81380f, 0.17610f, 0.60410f};
  // float correct[4] = {0.74f, 1.1192f, 0.3564f, 0.8744f};
  float *weights[1];
  float weight[8] = {0.15f, 0.1f, 0.19f, 0.35f, 0.4f, 0.54f, 0.42f, 0.68f};
  weights[0] = weight;
  float *biases[1];
  float bias[4] = {0.42f, 0.72f, 0.01f, 0.3f};
  biases[0] = bias;
  NeuralNetwork* model = buildModel(1, layers, weights, biases, 1.0, 1, 1);
  float input[4] = {0.32f, 0.68f, 0.83f, 0.02f};
  float *d_weights;
  float *d_input;
  float *d_product;
  float *d_bias;
  hipMalloc(&d_bias, model->layer_size[1]*sizeof(float));
  hipMemcpy(d_bias, model->biases[0], model->layer_size[1]*sizeof(float), hipMemcpyHostToDevice);
  hipMalloc(&d_weights, model->layer_size[0]*model->layer_size[1]*sizeof(float));
  hipMemcpy(d_weights, model->weights[0], model->layer_size[0]*model->layer_size[1]*sizeof(float), hipMemcpyHostToDevice);
  hipMalloc(&d_input, 4*sizeof(float));
  hipMemcpy(d_input, input, 4*sizeof(float), hipMemcpyHostToDevice);
  hipMalloc(&d_product, 8*sizeof(float));
  dim3 nBlocks(nWorkers, 1, 1);
  dim3 nThreads(nThreadsPerWorker, 1, 1);
  dotProductSegmented<<<nBlocks, nThreads>>>(d_input, d_weights, d_product, 2, model->layer_size[0], model->layer_size[0], model->layer_size[1], d_bias);
  hipDeviceSynchronize();
  float *prod = (float*)malloc(8*sizeof(float));
  hipMemcpy(prod, d_product, 8*sizeof(float), hipMemcpyDeviceToHost);
  for(int i = 0; i < 8; i++) {
    EXPECT_FLOAT_EQ(prod[i], correct[i]);
  }
}

TEST(SegmentedSigmoid, SingleThreadedEx1) {
  int nWorkers = 1;
  int nThreadsPerWorker = 1;
  int layers[2] = {1, 2};
  float correct[4] = {0.601807f, 0.58078581f, 0.6087355f, 0.59483749f};
  float *weights[1];
  weights[0] = (float*)malloc(2*sizeof(float));
  weights[0][0] = 0.1f;
  weights[0][1] = 0.2f;
  float *biases[1];
  biases[0] = (float*)malloc(2*sizeof(float));
  biases[0][0] = 0.4f;
  biases[0][1] = 0.3f;
  NeuralNetwork* model = buildModel(1, layers, weights, biases, 1.0, 1, 1);
  float input[2] = {0.13f, 0.42f};
  float product[4] = {1.0f, 1.0f, 1.0f, 1.0f};
  float *d_weights;
  float *d_input;
  float *d_product;
  float *d_bias;
  hipMalloc(&d_bias, 2*sizeof(float));
  hipMemcpy(d_bias, model->biases[0], 2*sizeof(float), hipMemcpyHostToDevice);
  hipMalloc(&d_weights, 2*sizeof(float));
  hipMemcpy(d_weights, model->weights[0], 2*sizeof(float), hipMemcpyHostToDevice);
  hipMalloc(&d_input, 2*sizeof(float));
  hipMemcpy(d_input, input, 2*sizeof(float), hipMemcpyHostToDevice);
  hipMalloc(&d_product, 4*sizeof(float));
  dim3 nBlocks(nWorkers, 1, 1);
  dim3 nThreads(nThreadsPerWorker, 1, 1);
  dotProductSegmented<<<nBlocks, nThreads>>>(d_input, d_weights, d_product, 2, model->layer_size[0], model->layer_size[0], model->layer_size[1], d_bias);
  hipDeviceSynchronize();
  sigmoidSegmented<<<nWorkers, nThreadsPerWorker>>>(d_product, 2*model->layer_size[1]);
  hipDeviceSynchronize();
  float *prod = (float*)malloc(4*sizeof(float));
  hipMemcpy(prod, d_product, 4*sizeof(float), hipMemcpyDeviceToHost);
  for(int i = 0; i < 4; i++) {
    EXPECT_FLOAT_EQ(prod[i], correct[i]);
  }
}

TEST(SegmentedSigmoid, SingleThreadedEx2) {
  int nWorkers = 1;
  int nThreadsPerWorker = 1;
  int layers[2] = {2, 4};
  float correct[8] = {0.67699581f, 0.75384f, 0.58816868f, 0.7056604f, 0.63471538f, 0.69291866f, 0.54391158f, 0.64659375f};
  // float correct[4] = {0.74f, 1.1192f, 0.3564f, 0.8744f};
  float *weights[1];
  float weight[8] = {0.15f, 0.1f, 0.19f, 0.35f, 0.4f, 0.54f, 0.42f, 0.68f};
  weights[0] = weight;
  float *biases[1];
  float bias[4] = {0.42f, 0.72f, 0.01f, 0.3f};
  biases[0] = bias;
  NeuralNetwork* model = buildModel(1, layers, weights, biases, 1.0, 1, 1);
  float input[4] = {0.32f, 0.68f, 0.83f, 0.02f};
  float *d_weights;
  float *d_input;
  float *d_product;
  float *d_bias;
  hipMalloc(&d_bias, model->layer_size[1]*sizeof(float));
  hipMemcpy(d_bias, model->biases[0], model->layer_size[1]*sizeof(float), hipMemcpyHostToDevice);
  hipMalloc(&d_weights, model->layer_size[0]*model->layer_size[1]*sizeof(float));
  hipMemcpy(d_weights, model->weights[0], model->layer_size[0]*model->layer_size[1]*sizeof(float), hipMemcpyHostToDevice);
  hipMalloc(&d_input, 4*sizeof(float));
  hipMemcpy(d_input, input, 4*sizeof(float), hipMemcpyHostToDevice);
  hipMalloc(&d_product, 8*sizeof(float));
  dim3 nBlocks(nWorkers, 1, 1);
  dim3 nThreads(nThreadsPerWorker, 1, 1);
  dotProductSegmented<<<nBlocks, nThreads>>>(d_input, d_weights, d_product, 2, model->layer_size[0], model->layer_size[0], model->layer_size[1], d_bias);
  hipDeviceSynchronize();
  sigmoidSegmented<<<nWorkers, nThreadsPerWorker>>>(d_product, 2*model->layer_size[1]);
  hipDeviceSynchronize();
  float *prod = (float*)malloc(8*sizeof(float));
  hipMemcpy(prod, d_product, 8*sizeof(float), hipMemcpyDeviceToHost);
  for(int i = 0; i < 8; i++) {
    EXPECT_FLOAT_EQ(prod[i], correct[i]);
  }
}

TEST(ForwardPass, SingleThreadedDotProduct2Ex1_BATCH_SIZE_1) {
  int nWorkers = 1;
  int nThreadsPerWorker = 1;
  int batch_size = 1;
  float correctOutput[6] = {0.601807f, 0.58078581f, 1.349375f, 0.6087355f, 0.59483749f, 1.3612702f};
  float input[2] = {0.13000f, 0.42f};
  int layers[3] = {1, 2, 1};
  float *weights[2];
  float weight0[2] = {0.1f, 0.2f};
  float weight1[2] = {0.5f, 0.6f};
  weights[0] = weight0;
  weights[1] = weight1;
  float *biases[2];
  float bias0[2] = {0.4f, 0.3f};
  float bias1[1] = {0.7f};
  biases[0] = bias0;
  biases[1] = bias1;
  NeuralNetwork* model = buildModel(2, layers, weights, biases, 1.0, nWorkers, nThreadsPerWorker);
  float *d_weights0;
  float *d_weights1;
  float *d_input;
  float *d_bias0;
  float *d_bias1;
  hipMalloc(&d_bias0, model->layer_size[1]*sizeof(float));
  hipMemcpy(d_bias0, model->biases[0], model->layer_size[1]*sizeof(float), hipMemcpyHostToDevice);
  hipMalloc(&d_bias1, model->layer_size[2]*sizeof(float));
  hipMemcpy(d_bias1, model->biases[1], model->layer_size[2]*sizeof(float), hipMemcpyHostToDevice);
  hipMalloc(&d_weights0, model->layer_size[0]*model->layer_size[1]*sizeof(float));
  hipMemcpy(d_weights0, model->weights[0], model->layer_size[0]*model->layer_size[1]*sizeof(float), hipMemcpyHostToDevice);
  hipMalloc(&d_weights1, model->layer_size[1]*model->layer_size[2]*sizeof(float));
  hipMemcpy(d_weights1, model->weights[1], model->layer_size[1]*model->layer_size[2]*sizeof(float), hipMemcpyHostToDevice);
  hipMalloc(&d_input, 2*sizeof(float));
  hipMemcpy(d_input, input, 2*sizeof(float), hipMemcpyHostToDevice);
  int activations_size = 0;
  int * offsets = new int[model->nLayers];
  for(int i = 1; i <= model->nLayers; i++) {
    offsets[i-1] = (batch_size * activations_size);
    // printf("Offset at %d: %d\n", i-1, offsets[i-1]);
    activations_size += model->layer_size[i];
  }
  EXPECT_EQ(offsets[0], 0);
  EXPECT_EQ(offsets[1], 2);
  float * d_activations = new float[batch_size*activations_size];
  float * activations = new float[batch_size*activations_size];
  //device pointers
  int * d_offsets;
  hipMalloc(&d_activations, activations_size*batch_size*sizeof(float));
  hipMalloc(&d_offsets, model->nLayers*sizeof(int));
  for(int i = 0; i < activations_size*batch_size; i++) {
    activations[i] = 1;
  }
  hipMemcpy(d_activations, activations, activations_size*batch_size*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_offsets, offsets, model->nLayers*sizeof(int), hipMemcpyHostToDevice);
  dim3 nBlocks(nWorkers, 1, 1);
  dim3 nThreads(nThreadsPerWorker, 1, 1);
  for(int i = 0; i < 2; i+=1) {
    dotProductSegmented<<<nBlocks, nThreads>>>(d_input+(i*model->layer_size[0]), d_weights0, d_activations, batch_size, model->layer_size[0], model->layer_size[0], model->layer_size[1], d_bias0);
    sigmoidSegmented<<<nWorkers, nThreadsPerWorker>>>(d_activations, batch_size*model->layer_size[1]);
    dotProductSegmented<<<nBlocks, nThreads>>>(d_activations, d_weights1, d_activations+(offsets[1]*batch_size), batch_size, model->layer_size[1], model->layer_size[1], model->layer_size[2], d_bias1);
    hipMemcpy(activations, d_activations, activations_size*batch_size*sizeof(float), hipMemcpyDeviceToHost);
    for(int j = 0; j < activations_size; j++) {
        EXPECT_FLOAT_EQ(correctOutput[i*activations_size+j], activations[j]);
    }
  }
}

TEST(ForwardPass, SingleThreadedDotProduct2Ex1_BATCH_SIZE_2) {
  int nWorkers = 1;
  int nThreadsPerWorker = 1;
  int batch_size = 2;
  float correctOutput[6] = {0.601807f, 0.58078581f, 0.6087355f, 0.59483749f, 1.349375f, 1.3612702f};
  float input[2] = {0.13000f, 0.42f};
  int layers[3] = {1, 2, 1};
  float *weights[2];
  float weight0[2] = {0.1f, 0.2f};
  float weight1[2] = {0.5f, 0.6f};
  weights[0] = weight0;
  weights[1] = weight1;
  float *biases[2];
  float bias0[2] = {0.4f, 0.3f};
  float bias1[1] = {0.7f};
  biases[0] = bias0;
  biases[1] = bias1;
  NeuralNetwork* model = buildModel(2, layers, weights, biases, 1.0, nWorkers, nThreadsPerWorker);
  float *d_weights0;
  float *d_weights1;
  float *d_input;
  float *d_bias0;
  float *d_bias1;
  hipMalloc(&d_bias0, model->layer_size[1]*sizeof(float));
  hipMemcpy(d_bias0, model->biases[0], model->layer_size[1]*sizeof(float), hipMemcpyHostToDevice);
  hipMalloc(&d_bias1, model->layer_size[2]*sizeof(float));
  hipMemcpy(d_bias1, model->biases[1], model->layer_size[2]*sizeof(float), hipMemcpyHostToDevice);
  hipMalloc(&d_weights0, model->layer_size[0]*model->layer_size[1]*sizeof(float));
  hipMemcpy(d_weights0, model->weights[0], model->layer_size[0]*model->layer_size[1]*sizeof(float), hipMemcpyHostToDevice);
  hipMalloc(&d_weights1, model->layer_size[1]*model->layer_size[2]*sizeof(float));
  hipMemcpy(d_weights1, model->weights[1], model->layer_size[1]*model->layer_size[2]*sizeof(float), hipMemcpyHostToDevice);
  hipMalloc(&d_input, 2*sizeof(float));
  hipMemcpy(d_input, input, 2*sizeof(float), hipMemcpyHostToDevice);
  int activations_size = 0;
  int * offsets = new int[model->nLayers];
  for(int i = 1; i <= model->nLayers; i++) {
    offsets[i-1] = (batch_size * activations_size);
    // printf("Offset at %d: %d\n", i-1, offsets[i-1]);
    activations_size += model->layer_size[i];
  }
  EXPECT_EQ(offsets[0], 0);
  EXPECT_EQ(offsets[1], 4);
  EXPECT_EQ(activations_size*batch_size, 6);
  float * d_activations = new float[batch_size*activations_size];
  float * activations = new float[batch_size*activations_size];
  //device pointers
  int * d_offsets;
  hipMalloc(&d_activations, activations_size*batch_size*sizeof(float));
  hipMalloc(&d_offsets, model->nLayers*sizeof(int));
  for(int i = 0; i < activations_size*batch_size; i++) {
    activations[i] = 1;
  }
  hipMemcpy(d_activations, activations, activations_size*batch_size*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_offsets, offsets, model->nLayers*sizeof(int), hipMemcpyHostToDevice);
  dim3 nBlocks(nWorkers, 1, 1);
  dim3 nThreads(nThreadsPerWorker, 1, 1);
  for(int i = 0; i < 2; i+=batch_size) {
    dotProductSegmented<<<nBlocks, nThreads>>>(d_input+(i*model->layer_size[0]), d_weights0, d_activations, batch_size, model->layer_size[0], model->layer_size[0], model->layer_size[1], d_bias0);
    sigmoidSegmented<<<nWorkers, nThreadsPerWorker>>>(d_activations, batch_size*model->layer_size[1]);
    dotProductSegmented<<<nBlocks, nThreads>>>(d_activations, d_weights1, d_activations+(offsets[1]), batch_size, model->layer_size[1], model->layer_size[1], model->layer_size[2], d_bias1);
    hipMemcpy(activations, d_activations, activations_size*batch_size*sizeof(float), hipMemcpyDeviceToHost);
    for(int j = 0; j < activations_size*batch_size; j++) {
        printf("j: %d\n", j);
        EXPECT_FLOAT_EQ(correctOutput[i*activations_size+j], activations[j]);
    }
  }
}

TEST(ForwardPass, SingleThreadedSoftmaxEx1_BATCH_SIZE_1) {
  int nWorkers = 1;
  int nThreadsPerWorker = 1;
  int batch_size = 1;
  float input[2] = {0.13000f, 0.42f};
  int layers[3] = {1, 2, 1};
  float correctOutput[2] = {1.0f, 1.0f};
  float *weights[2];
  float weight0[2] = {0.1f, 0.2f};
  float weight1[2] = {0.5f, 0.6f};
  weights[0] = weight0;
  weights[1] = weight1;
  float *biases[2];
  float bias0[2] = {0.4f, 0.3f};
  float bias1[1] = {0.7f};
  biases[0] = bias0;
  biases[1] = bias1;
  NeuralNetwork* model = buildModel(2, layers, weights, biases, 1.0, nWorkers, nThreadsPerWorker);
  float *d_weights0;
  float *d_weights1;
  float *d_input;
  float *d_bias0;
  float *d_bias1;
  hipMalloc(&d_bias0, model->layer_size[1]*sizeof(float));
  hipMemcpy(d_bias0, model->biases[0], model->layer_size[1]*sizeof(float), hipMemcpyHostToDevice);
  hipMalloc(&d_bias1, model->layer_size[2]*sizeof(float));
  hipMemcpy(d_bias1, model->biases[1], model->layer_size[2]*sizeof(float), hipMemcpyHostToDevice);
  hipMalloc(&d_weights0, model->layer_size[0]*model->layer_size[1]*sizeof(float));
  hipMemcpy(d_weights0, model->weights[0], model->layer_size[0]*model->layer_size[1]*sizeof(float), hipMemcpyHostToDevice);
  hipMalloc(&d_weights1, model->layer_size[1]*model->layer_size[2]*sizeof(float));
  hipMemcpy(d_weights1, model->weights[1], model->layer_size[1]*model->layer_size[2]*sizeof(float), hipMemcpyHostToDevice);
  hipMalloc(&d_input, 2*sizeof(float));
  hipMemcpy(d_input, input, 2*sizeof(float), hipMemcpyHostToDevice);
  int activations_size = 0;
  int * offsets = new int[model->nLayers];
  for(int i = 1; i <= model->nLayers; i++) {
    offsets[i-1] = (batch_size * activations_size);
    // printf("Offset at %d: %d\n", i-1, offsets[i-1]);
    activations_size += model->layer_size[i];
  }
  EXPECT_EQ(offsets[0], 0);
  EXPECT_EQ(offsets[1], 2);
  float * d_activations = new float[batch_size*activations_size];
  float * activations = new float[batch_size*activations_size];
  //device pointers
  int * d_offsets;
  hipMalloc(&d_activations, activations_size*batch_size*sizeof(float));
  hipMalloc(&d_offsets, model->nLayers*sizeof(int));
  for(int i = 0; i < activations_size*batch_size; i++) {
    activations[i] = -1;
  }
  hipMemcpy(d_activations, activations, activations_size*batch_size*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_offsets, offsets, model->nLayers*sizeof(int), hipMemcpyHostToDevice);
  dim3 nBlocks(nWorkers, 1, 1);
  dim3 nThreads(nThreadsPerWorker, 1, 1);
  for(int i = 0; i < 2; i+=batch_size) {
    dotProductSegmented<<<nBlocks, nThreads>>>(d_input+(i*model->layer_size[0]), d_weights0, d_activations, batch_size, model->layer_size[0], model->layer_size[0], model->layer_size[1], d_bias0);
    sigmoidSegmented<<<nWorkers, nThreadsPerWorker>>>(d_activations, batch_size*model->layer_size[1]);
    dotProductSegmented<<<nBlocks, nThreads>>>(d_activations, d_weights1, d_activations+(offsets[1]), batch_size, model->layer_size[1], model->layer_size[1], model->layer_size[2], d_bias1);
    softmaxSegmented<<<nWorkers, nThreadsPerWorker>>>(d_activations+(offsets[1]*batch_size), batch_size, model->layer_size[2]);
    hipMemcpy(activations, d_activations, activations_size*batch_size*sizeof(float), hipMemcpyDeviceToHost);
    EXPECT_FLOAT_EQ(correctOutput[i], activations[2]);
  }
}

TEST(ForwardPass, SingleThreadedSoftmaxEx1_BATCH_SIZE_2) {
  int nWorkers = 1;
  int nThreadsPerWorker = 1;
  int batch_size = 2;
  float input[2] = {0.13000f, 0.42f};
  int layers[3] = {1, 2, 1};
  float correctOutput[2] = {1.0f, 1.0f};
  float *weights[2];
  float weight0[2] = {0.1f, 0.2f};
  float weight1[2] = {0.5f, 0.6f};
  weights[0] = weight0;
  weights[1] = weight1;
  float *biases[2];
  float bias0[2] = {0.4f, 0.3f};
  float bias1[1] = {0.7f};
  biases[0] = bias0;
  biases[1] = bias1;
  NeuralNetwork* model = buildModel(2, layers, weights, biases, 1.0, nWorkers, nThreadsPerWorker);
  float *d_weights0;
  float *d_weights1;
  float *d_input;
  float *d_bias0;
  float *d_bias1;
  hipMalloc(&d_bias0, model->layer_size[1]*sizeof(float));
  hipMemcpy(d_bias0, model->biases[0], model->layer_size[1]*sizeof(float), hipMemcpyHostToDevice);
  hipMalloc(&d_bias1, model->layer_size[2]*sizeof(float));
  hipMemcpy(d_bias1, model->biases[1], model->layer_size[2]*sizeof(float), hipMemcpyHostToDevice);
  hipMalloc(&d_weights0, model->layer_size[0]*model->layer_size[1]*sizeof(float));
  hipMemcpy(d_weights0, model->weights[0], model->layer_size[0]*model->layer_size[1]*sizeof(float), hipMemcpyHostToDevice);
  hipMalloc(&d_weights1, model->layer_size[1]*model->layer_size[2]*sizeof(float));
  hipMemcpy(d_weights1, model->weights[1], model->layer_size[1]*model->layer_size[2]*sizeof(float), hipMemcpyHostToDevice);
  hipMalloc(&d_input, sizeof(float));
  hipMemcpy(d_input, input, sizeof(float), hipMemcpyHostToDevice);
  int activations_size = 0;
  int * offsets = new int[model->nLayers];
  for(int i = 1; i <= model->nLayers; i++) {
    offsets[i-1] = (batch_size * activations_size);
    // printf("Offset at %d: %d\n", i-1, offsets[i-1]);
    activations_size += model->layer_size[i];
  }
  EXPECT_EQ(offsets[0], 0);
  EXPECT_EQ(offsets[1], 4);
  float * d_activations = new float[batch_size*activations_size];
  float * activations = new float[batch_size*activations_size];
  //device pointers
  int * d_offsets;
  hipMalloc(&d_activations, activations_size*batch_size*sizeof(float));
  hipMalloc(&d_offsets, model->nLayers*sizeof(int));
  for(int i = 0; i < activations_size*batch_size; i++) {
    activations[i] = -1;
  }
  hipMemcpy(d_activations, activations, activations_size*batch_size*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_offsets, offsets, model->nLayers*sizeof(int), hipMemcpyHostToDevice);
  dim3 nBlocks(nWorkers, 1, 1);
  dim3 nThreads(nThreadsPerWorker, 1, 1);
  for(int i = 0; i < 2; i+=batch_size) {
    dotProductSegmented<<<nBlocks, nThreads>>>(d_input+(i*model->layer_size[0]), d_weights0, d_activations, batch_size, model->layer_size[0], model->layer_size[0], model->layer_size[1], d_bias0);
    sigmoidSegmented<<<nWorkers, nThreadsPerWorker>>>(d_activations, batch_size*model->layer_size[1]);
    dotProductSegmented<<<nBlocks, nThreads>>>(d_activations, d_weights1, d_activations+(offsets[1]), batch_size, model->layer_size[1], model->layer_size[1], model->layer_size[2], d_bias1);
    softmaxSegmented<<<nWorkers, nThreadsPerWorker>>>(d_activations+(offsets[1]), batch_size, model->layer_size[2]);
    hipMemcpy(activations, d_activations, activations_size*batch_size*sizeof(float), hipMemcpyDeviceToHost);
    for(int j = 0; j < batch_size; j++) {
        EXPECT_FLOAT_EQ(correctOutput[j], activations[offsets[1]+j]);
    }
  }
}
TEST(ForwardPass, SingleThreadedSoftmaxEx1_2) {
  int nWorkers = 1;
  int nThreadsPerWorker = 1;
  int batch_size = 1;
  float input[1] = {0.13000f};
  int layers[3] = {1, 2, 1};
  float *weights[2];
  float weight0[2] = {0.1f, 0.2f};
  float weight1[2] = {0.5f, 0.6f};
  weights[0] = weight0;
  weights[1] = weight1;
  float *biases[2];
  float bias0[2] = {0.4f, 0.3f};
  float bias1[1] = {0.7f};
  biases[0] = bias0;
  biases[1] = bias1;
  NeuralNetwork* model = buildModel(2, layers, weights, biases, 1.0, nWorkers, nThreadsPerWorker);
  float *d_weights0;
  float *d_weights1;
  float *d_input;
  float *d_bias0;
  float *d_bias1;
  hipMalloc(&d_bias0, model->layer_size[1]*sizeof(float));
  hipMemcpy(d_bias0, model->biases[0], model->layer_size[1]*sizeof(float), hipMemcpyHostToDevice);
  hipMalloc(&d_bias1, model->layer_size[2]*sizeof(float));
  hipMemcpy(d_bias1, model->biases[1], model->layer_size[2]*sizeof(float), hipMemcpyHostToDevice);
  hipMalloc(&d_weights0, model->layer_size[0]*model->layer_size[1]*sizeof(float));
  hipMemcpy(d_weights0, model->weights[0], model->layer_size[0]*model->layer_size[1]*sizeof(float), hipMemcpyHostToDevice);
  hipMalloc(&d_weights1, model->layer_size[1]*model->layer_size[2]*sizeof(float));
  hipMemcpy(d_weights1, model->weights[1], model->layer_size[1]*model->layer_size[2]*sizeof(float), hipMemcpyHostToDevice);
  hipMalloc(&d_input, sizeof(float));
  hipMemcpy(d_input, input, sizeof(float), hipMemcpyHostToDevice);
  int activations_size = 0;
  int * offsets = new int[model->nLayers];
  for(int i = 1; i <= model->nLayers; i++) {
    offsets[i-1] = (batch_size * activations_size);
    // printf("Offset at %d: %d\n", i-1, offsets[i-1]);
    activations_size += model->layer_size[i];
  }
  EXPECT_EQ(offsets[0], 0);
  EXPECT_EQ(offsets[1], 2);
  float * d_activations = new float[batch_size*activations_size];
  float * activations = new float[batch_size*activations_size];
  //device pointers
  int * d_offsets;
  hipMalloc(&d_activations, activations_size*batch_size*sizeof(float));
  hipMalloc(&d_offsets, model->nLayers*sizeof(int));
  for(int i = 0; i < activations_size*batch_size; i++) {
    activations[i] = 1;
  }
  hipMemcpy(d_activations, activations, activations_size*batch_size*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_offsets, offsets, model->nLayers*sizeof(int), hipMemcpyHostToDevice);
  dim3 nBlocks(nWorkers, 1, 1);
  dim3 nThreads(nThreadsPerWorker, 1, 1);
  dotProductSegmented<<<nBlocks, nThreads>>>(d_input, d_weights0, d_activations, batch_size, model->layer_size[0], model->layer_size[0], model->layer_size[1], d_bias0);
  sigmoidSegmented<<<nWorkers, nThreadsPerWorker>>>(d_activations, batch_size*model->layer_size[1]);
  dotProductSegmented<<<nBlocks, nThreads>>>(d_activations, d_weights1, d_activations+(offsets[1]*batch_size), batch_size, model->layer_size[1], model->layer_size[1], model->layer_size[2], d_bias1);
  softmaxSegmented<<<nWorkers, nThreadsPerWorker>>>(d_activations+(offsets[1]*batch_size), batch_size, model->layer_size[2]);
  float correctOutput[3] = {0.601807f, 0.58078581f, 1.0f};
  hipMemcpy(activations, d_activations, activations_size*batch_size*sizeof(float), hipMemcpyDeviceToHost);
  for(int i = 0; i < activations_size; i++) {
     EXPECT_FLOAT_EQ(correctOutput[i], activations[i]);
  }
}